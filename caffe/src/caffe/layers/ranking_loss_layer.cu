#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/ranking_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void RankingLossForwardGPU(const int nthreads,
          const Dtype* bottom_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, Dtype margin) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / dim;    //row
    const int label_value = static_cast<int>(label[n]);
    const int label_index = n*dim + label_value;
    if( label_index != index ) {
     loss[index] = max( Dtype(0),
	margin - bottom_data[label_index] + bottom_data[index] );
    } else {
     loss[index] = 0;
    }
  }
}
template <typename Dtype>
void RankingLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int num = bottom[0]->num();
  const int dim = bottom[0]->count()/num;
  const int nthreads = num * dim; 
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  RankingLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_data, label, loss_data,
      num, dim , margin); 
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  loss /= num;
  top[0]->mutable_cpu_data()[0] = loss;
}


//
//implementation of 1D Grid and 1D Blocks 
// TODO( measure cpu_time and improve performace)
template <typename Dtype>
__global__ void RankingLossBackwardGPU(  const Dtype* bottom_data,const Dtype* label, 
                    Dtype* bottom_diff, const int num, const int dim, int trunc, Dtype margin) {
 unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  if(ix < num) { 
  // printf("%d %d %d\n",ix,trunc,dim);
   const int label_value = static_cast<int>(label[ix]); 
   const int label_index = dim*ix + label_value;
   for( int iy = 0 ; iy < dim ; iy++){
	const int index = dim*ix + iy;
	if( label_index != index ) {
         const Dtype prob = max( Dtype(0),
	  margin - bottom_data[label_index] + bottom_data[index] );
	 if( prob > 0){
	  bottom_diff[index] = 1;
	  bottom_diff[label_index] -= 1;
	  if(trunc) break; //this is for truncate version
	}
      }
    }     
  }
}
template <typename Dtype>
void RankingLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    const int Trunc_ = this->layer_param_.ranking_loss_param().trunc();
    const int num = bottom[0]->num();
    const int dim = bottom[0]->count() / num;
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype scale = top[0]->cpu_diff()[0]/num;
    //initalize bottom_diff to zero
    caffe_gpu_set(bottom[0]->count() , Dtype(0) , bottom_diff);
    const Dtype* label = bottom[1]->gpu_data();
    //Checking most powerful performance
    dim3 block(32); dim3 grid((num + block.x - 1) / block.x);
    RankingLossBackwardGPU<Dtype><<<grid,
         block>>>( bottom_data, label, bottom_diff,
         num, dim, Trunc_,margin); 
    caffe_gpu_scal(bottom[0]->count(), scale , bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(RankingLossLayer);

}  // namespace caffe
